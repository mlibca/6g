
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <hipfft/hipfft.h>
#include <stdlib.h>
#include <iostream>

// CUDA kernel function (device code)
__global__ void cudakernel(int *a) {
    int idx = threadIdx.x;
    a[idx] = idx;
   

}

int main() {
    const int N = 2560;
    int h_a[N];

    // Allocate memory on the device
    int *d_a;
    hipMalloc((void **)&d_a, N * sizeof(int));

    // Launch the kernel
    cudakernel<<<1, N>>>(d_a);

    // Copy result from device to host
    hipMemcpy(h_a, d_a, N * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < N; i++) {
        std::cout << h_a[i] << " ";
    }
    std::cout << std::endl;

    // Free memory
    hipFree(d_a);

    return 0;
}
